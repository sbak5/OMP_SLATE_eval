#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// Copyright (c) 2017, University of Tennessee
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the University of Tennessee nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL UNIVERSITY OF TENNESSEE BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//------------------------------------------------------------------------------
// This research was supported by the Exascale Computing Project (17-SC-20-SC),
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.
//------------------------------------------------------------------------------
// For assistance with SLATE, email <slate-user@icl.utk.edu>.
// You can also join the "SLATE User" Google group by going to
// https://groups.google.com/a/icl.utk.edu/forum/#!forum/slate-user,
// signing in with your Google credentials, and then clicking "Join group".
//------------------------------------------------------------------------------

#include "slate/internal/device.hh"
#include "device_util.cuh"

#include <cstdio>
#include <hip/hip_complex.h>

namespace slate {
namespace device {

//------------------------------------------------------------------------------
/// Kernel implementing element-wise tile set.
/// Each thread block deals with one tile.
/// Each thread deals with one row.
/// Launched by geset().
///
/// @param[in] m
///     Number of rows of each tile. m >= 1.
///
/// @param[in] n
///     Number of columns of each tile. n >= 1.
///
/// @param[in] offdiag_value
///     The value to set outside of the diagonal.
///
/// @param[in] diag_value
///     The value to set on the diagonal.
///
/// @param[in] Atiles
///     Array of tiles of dimension gridDim.x,
///     where each Atiles[k] is an m-by-n matrix stored in an lda-by-n array.
///
/// @param[in] lda
///     Leading dimension of each tile in Atiles. lda >= m.
///
template <typename scalar_t>
__global__ void gesetKernel(
    int64_t m, int64_t n,
    scalar_t offdiag_value, scalar_t diag_value, scalar_t** tilesA, int64_t lda)
{
    scalar_t* tileA = tilesA[blockIdx.x];

    // thread per row, if more rows than threads, loop by blockDim.x
    for (int64_t ridx = threadIdx.x; ridx < m; ridx += blockDim.x) {
        // todo: should the increment be ridx += 1024?
        scalar_t* rowA = &tileA[ridx];

        for (int64_t j = 0; j < n; ++j)
            rowA[j*lda] = (j != ridx) ? offdiag_value : diag_value;
    }
}

//------------------------------------------------------------------------------
/// Batched routine for element-wise tile set.
///
/// @param[in] m
///     Number of rows of each tile. m >= 0.
///
/// @param[in] n
///     Number of columns of each tile. n >= 0.
///
/// @param[in] diag_value
///     The value to set on the diagonal.
///
/// @param[in] offdiag_value
///     The value to set outside of the diagonal.
///
/// @param[in] Aarray
///     Array in GPU memory of dimension batch_count, containing pointers to tiles,
///     where each Aarray[k] is an m-by-n matrix stored in an lda-by-n array in GPU memory.
///
/// @param[in] lda
///     Leading dimension of each tile in A. lda >= m.
///
/// @param[in] batch_count
///     Size of Aarray and Barray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void geset(
    int64_t m, int64_t n,
    scalar_t diag_value, scalar_t offdiag_value, scalar_t** Aarray, int64_t lda,
    int64_t batch_count, hipStream_t stream)
{
    // quick return
    if (batch_count == 0)
        return;

    // Max threads/block=1024 for current CUDA compute capability (<=7.5)
    int64_t nthreads = std::min((int64_t)1024 , m);

    gesetKernel<<<batch_count, nthreads, 0, stream>>>(
        m, n,
        diag_value, offdiag_value, Aarray, lda);

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
// Explicit instantiations.
template
void geset(
    int64_t m, int64_t n,
    float diag_value, float offdiag_value, float** Aarray, int64_t lda,
    int64_t batch_count, hipStream_t stream);

template
void geset(
    int64_t m, int64_t n,
    double diag_value, double offdiag_value, double** Aarray, int64_t lda,
    int64_t batch_count, hipStream_t stream);

template
void geset(
    int64_t m, int64_t n,
    hipFloatComplex diag_value, hipFloatComplex offdiag_value,
    hipFloatComplex** Aarray, int64_t lda,
    int64_t batch_count, hipStream_t stream);

template
void geset(
    int64_t m, int64_t n,
    hipDoubleComplex diag_value, hipDoubleComplex offdiag_value,
    hipDoubleComplex** Aarray, int64_t lda,
    int64_t batch_count, hipStream_t stream);

} // namespace device
} // namespace slate
