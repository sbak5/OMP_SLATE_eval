#include "hip/hip_runtime.h"
//------------------------------------------------------------------------------
// Copyright (c) 2017, University of Tennessee
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the University of Tennessee nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL UNIVERSITY OF TENNESSEE BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//------------------------------------------------------------------------------
// This research was supported by the Exascale Computing Project (17-SC-20-SC),
// a collaborative effort of two U.S. Department of Energy organizations (Office
// of Science and the National Nuclear Security Administration) responsible for
// the planning and preparation of a capable exascale ecosystem, including
// software, applications, hardware, advanced system engineering and early
// testbed platforms, in support of the nation's exascale computing imperative.
//------------------------------------------------------------------------------
// For assistance with SLATE, email <slate-user@icl.utk.edu>.
// You can also join the "SLATE User" Google group by going to
// https://groups.google.com/a/icl.utk.edu/forum/#!forum/slate-user,
// signing in with your Google credentials, and then clicking "Join group".
//------------------------------------------------------------------------------

#include "slate/internal/device.hh"
#include "device_util.cuh"

#include <cstdio>
#include <hip/hip_complex.h>

namespace slate {
namespace device {

// internal blocking
// 16 x 16 thread block = 256 threads
// 32 x 32 thread block = 1024 threads
static const int ib = 16;

//------------------------------------------------------------------------------
/// Device routine handles one matrix.
/// Thread block grid:
/// x = batch index (ignored here; see batch kernel),
/// y = block row index,
/// z = block col index.
/// Each thread block is ib-by-ib threads and does one ib-by-ib block of an
/// n-by-n matrix.
///
/// Let nt = ceildiv( n, ib ) be the number of blocks for one n-by-n matrix.
/// An even number of blocks uses an (nt + 1) by (nt/2) grid.
/// Example: for nt = 4 blocks, y by z = 5 by 2 grid:
///   [ A00  A01 ]
///   [----. A11 ]                  [ A10  .  |  .   .  ]
///   [ A10 '----]                  [ A20 A21 |  .   .  ]
///   [ A20  A21 ] covers matrix as [ A30 A31 | A00  .  ]
///   [ A30  A31 ]                  [ A40 A41 | A01 A11 ]
///   [ A40  A41 ]
///
/// An odd number of blocks uses an (nt) by (nt + 1)/2 grid.
/// Example: for nt = 5 blocks, y by z = 5 by 3 grid:
///   [ A00 | A01   A02 ]
///   [     '----.      ]                  [ A00  .   .  |  .   .  ]
///   [ A10   A11 | A12 ]                  [ A10 A11  .  |  .   .  ]
///   [           '-----] covers matrix as [ A20 A21 A22 |  .   .  ]
///   [ A20   A21   A22 ]                  [ A30 A31 A32 | A01  .  ]
///   [ A30   A31   A32 ]                  [ A40 A41 A42 | A02 A12 ]
///   [ A40   A41   A42 ]
///
template <typename scalar_t>
__device__ void transpose_func(
    int n,
    scalar_t* A, int64_t lda)
{
    // +1 to avoid memory bank conflicts.
    __shared__ scalar_t sA1[ ib ][ ib+1 ];
    __shared__ scalar_t sA2[ ib ][ ib+1 ];

    // i, j are row & column indices of top-left corner of each block.
    // ii, jj are row & column offsets within each block.
    int ii = threadIdx.x;
    int jj = threadIdx.y;

    int i, j;
    if (gridDim.y - 1 == gridDim.z*2) {
        // Even number of blocks.
        //assert( ceildiv(n, ib) % 2 == 0 );
        bool lower = (blockIdx.y > blockIdx.z);
        i = (lower ? (blockIdx.y - 1) : (blockIdx.z + gridDim.z));
        j = (lower ? (blockIdx.z    ) : (blockIdx.y + gridDim.z));
    }
    else {
        // Odd number of blocks.
        //assert( ceildiv(n, ib) % 2 == 1 );
        bool lower = (blockIdx.y >= blockIdx.z);
        i = (lower ? blockIdx.y : (blockIdx.z + gridDim.z - 1));
        j = (lower ? blockIdx.z : (blockIdx.y + gridDim.z    ));
    }
    i *= ib;
    j *= ib;

    scalar_t* A1 = A + i + ii + (j + jj)*lda;  // A(i, j)
    if (i == j) { // diagonal block
        // Load block A(i, j) into shared memory sA1.
        if (i + ii < n  &&  j + jj < n) {
            sA1[jj][ii] = *A1;
        }
        __syncthreads();

        // Save transposed block, A(i, j) = trans(sA1).
        if (i + ii < n  &&  j + jj < n) {
            *A1 = sA1[ii][jj];
        }
    }
    else { // off-diagonal block
        scalar_t* A2 = A + j + ii + (i + jj)*lda;  // A(j, i)
        // Load blocks A(i, j) and A(j, i) into shared memory sA1 and sA2.
        if (i + ii < n  &&  j + jj < n) {
            sA1[jj][ii] = *A1;
        }
        if (j + ii < n  &&  i + jj < n) {
            sA2[jj][ii] = *A2;
        }
        __syncthreads();

        // Save transposed blocks, A(i, j) = trans(sA2), A(j, i) = trans(sA1).
        if (i + ii < n && j + jj < n) {
            *A1 = sA2[ii][jj];
        }
        if (j + ii < n && i + jj < n) {
            *A2 = sA1[ii][jj];
        }
    }
}

//------------------------------------------------------------------------------

static const int NB = 32;
static const int NY = 8;
// static const int NX = 32; handled as template parameter, look below


/// tile M-by-N matrix with ceil(M/NB) by ceil(N/NB) tiles sized NB-by-NB.
/// uses NX-by-NY threads, where NB/NX, NB/NY, NX/NY evenly.
/// subtile each NB-by-NB tile with (NB/NX) subtiles sized NX-by-NB
/// for each subtile
///     load NX-by-NB subtile transposed from A into sA, as (NB/NY) blocks sized NX-by-NY
///     save NB-by-NX subtile from sA into AT,   as (NB/NX)*(NX/NY) blocks sized NX-by-NY
///     A  += NX
///     AT += NX*ldat
///
/// e.g., with NB=32, NX=32, NY=8 ([sdc] precisions)
///     load 32x32 subtile as 4   blocks of 32x8 columns: (A11  A12  A13  A14 )
///     save 32x32 subtile as 1*4 blocks of 32x8 columns: (AT11 AT12 AT13 AT14)
///
/// e.g., with NB=32, NX=16, NY=8 (z precision)
///     load 16x32 subtile as 4   blocks of 16x8 columns: (A11  A12  A13  A14)
///     save 32x16 subtile as 2*2 blocks of 16x8 columns: (AT11 AT12)
///                                                       (AT21 AT22)
///
template <typename scalar_t, int NX>
__device__ void transpose_func(
    int m, int n,
    const scalar_t *A,  int64_t lda,
          scalar_t *AT, int64_t ldat)
{
    __shared__ scalar_t sA[NB][NX+1];

    int tx  = threadIdx.x;
    int ty  = threadIdx.y;
    int iby = blockIdx.y*NB;
    int ibz = blockIdx.z*NB;
    int i, j;

    A  += iby + tx + (ibz + ty)*lda;
    AT += ibz + tx + (iby + ty)*ldat;

    #pragma unroll
    for (int tile=0; tile < NB/NX; ++tile) {
        // load NX-by-NB subtile transposed from A into sA
        i = iby + tx + tile*NX;
        j = ibz + ty;
        if (i < m) {
            #pragma unroll
            for (int j2=0; j2 < NB; j2 += NY) {
                if (j + j2 < n) {
                    sA[ty + j2][tx] = A[j2*lda];
                }
            }
        }
        __syncthreads();

        // save NB-by-NX subtile from sA into AT
        i = ibz + tx;
        j = iby + ty + tile*NX;
        #pragma unroll
        for (int i2=0; i2 < NB; i2 += NX) {
            if (i + i2 < n) {
                #pragma unroll
                for (int j2=0; j2 < NX; j2 += NY) {
                    if (j + j2 < m) {
                        AT[i2 + j2*ldat] = sA[tx + i2][ty + j2];
                    }
                }
            }
        }
        __syncthreads();

        // move to next subtile
        A  += NX;
        AT += NX*ldat;
    }
}

//------------------------------------------------------------------------------
/// in-place transpose of a square buffer
template <typename scalar_t>
__global__ void transpose_kernel(
    int n,
    scalar_t* A, int64_t lda)
{
    transpose_func(n, A, lda);
}

//------------------------------------------------------------------------------
/// in-place transpose of array of square buffers
template <typename scalar_t>
__global__ void transpose_batch_kernel(
    int n,
    scalar_t** Aarray, int64_t lda)
{
    transpose_func(n, Aarray[blockIdx.x], lda);
}

//------------------------------------------------------------------------------
/// out-of-place transpose of a rectangular buffer
/// transopses A onto AT
///
template <typename scalar_t, int NX>
__global__ void transpose_kernel(
    int m, int n,
    const scalar_t *A,  int64_t lda,
          scalar_t *AT, int64_t ldat)
{
    transpose_func<scalar_t, NX>(m, n, A, lda, AT, ldat);
}

//------------------------------------------------------------------------------
/// out-of-place transpose of an array of rectangular buffers
/// transopses dA_array onto dAT_array
///
template <typename scalar_t, int NX>
__global__ void transpose_batch_kernel(
    int m, int n,
    scalar_t **dA_array,  int64_t lda,
    scalar_t **dAT_array, int64_t ldat)
{
    transpose_func<scalar_t, NX>(m, n, dA_array[blockIdx.x], lda, dAT_array[blockIdx.x], ldat);
}

//------------------------------------------------------------------------------
/// Physically transpose a square matrix in place.
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 0.
///
/// @param[in,out] A
///     A square n-by-n matrix stored in an lda-by-n array in GPU memory.
///     On output, A is transposed.
///
/// @param[in] lda
///     Leading dimension of A. lda >= n.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void transpose(
    int64_t n,
    scalar_t* A, int64_t lda,
    hipStream_t stream)
{
    if (n <= 1)
        return;
    assert(lda >= n);

    int nt = ceildiv( n, int64_t(ib) );
    assert(nt <= 65535);                // CUDA limitation

    // Need 1/2 * (nt + 1) * nt to cover lower triangle and diagonal of matrix.
    // Block assignment differs depending on whether nt is odd or even.
    dim3 blocks;
    if (nt % 2 == 0) {
        // even blocks
        blocks = { 1, uint(nt + 1), uint(nt/2) };
    }
    else {
        // odd blocks
        blocks = { 1, uint(nt), uint((nt + 1)/2) };
    }
    dim3 threads( ib, ib );

    transpose_kernel<<< blocks, threads, 0, stream >>>
        (n, A, lda);

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
/// Physically transpose a batch of square matrices in place.
///
/// @param[in] n
///     Number of rows and columns of each tile. n >= 0.
///
/// @param[in,out] Aarray
///     Array in GPU memory of dimension batch_count, containing pointers to
///     matrices, where each Aarray[k] is a square n-by-n matrix stored in an
///     lda-by-n array in GPU memory.
///     On output, each Aarray[k] is transposed.
///
/// @param[in] lda
///     Leading dimension of each tile. lda >= n.
///
/// @param[in] batch_count
///     Size of Aarray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t>
void transpose_batch(
    int64_t n,
    scalar_t** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream)
{
    if (batch_count < 0 || n <= 1)
        return;
    assert(lda >= n);

    int nt = ceildiv( n, int64_t(ib) );
    assert(nt <= 65535);                // CUDA limitation
    assert(batch_count <= 2147483647);  // CUDA limitation, 2^31 - 1

    // Need 1/2 * (nt + 1) * nt to cover lower triangle and diagonal of matrix.
    // Block assignment differs depending on whether nt is odd or even.
    dim3 blocks;
    if (nt % 2 == 0) {
        // even blocks
        blocks = { uint(batch_count), uint(nt + 1), uint(nt/2) };
    }
    else {
        // odd blocks
        blocks = { uint(batch_count), uint(nt), uint((nt + 1)/2) };
    }
    dim3 threads( ib, ib );

    transpose_batch_kernel<<< blocks, threads, 0, stream >>>
        (n, Aarray, lda);

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
/// Physically transpose a rectangular matrix out-of-place.
///
/// @param[in] m
///     Number of columns of tile. m >= 0.
///
/// @param[in] n
///     Number of rows of tile. n >= 0.
///
/// @param[in] dA
///     A rectangular m-by-n matrix stored in an lda-by-n array in GPU memory.
///
/// @param[in] lda
///     Leading dimension of dA. lda >= m.
///
/// @param[out] dAT
///     A rectangular m-by-n matrix stored in an ldat-by-m array in GPU memory.
///     On output, dAT is the transpose of dA.
///
/// @param[in] ldat
///     Leading dimension of dAT. ldat >= n.
///
/// @param[in] batch_count
///     Size of Aarray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t, int NX>
void transpose(
    int64_t m, int64_t n,
    scalar_t* dA,  int64_t lda,
    scalar_t* dAT, int64_t ldat,
    hipStream_t stream)
{
    if ((m <= 0) || (n <= 0))
        return;
    assert(lda >= m);
    assert(ldat >= n);

    int mt = ceildiv( m, int64_t(NB) );
    assert(mt <= 65535);                // CUDA limitation
    int nt = ceildiv( n, int64_t(NB) );
    assert(nt <= 65535);                // CUDA limitation

    dim3 grid( 1, mt, nt );
    dim3 threads( NX, NY );
    transpose_kernel<scalar_t, NX><<< grid, threads, 0, stream >>>
        ( m, n, dA, lda, dAT, ldat );

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
/// Physically transpose a batch of rectangular matrices out-of-place.
///
/// @param[in] m
///     Number of columns of each tile. m >= 0.
///
/// @param[in] n
///     Number of rows of each tile. n >= 0.
///
/// @param[in] dA_array
///     Array in GPU memory of dimension batch_count, containing pointers to
///     matrices, where each dA_array[k] is a rectangular m-by-n matrix stored in an
///     lda-by-n array in GPU memory.
///
/// @param[in] lda
///     Leading dimension of each dA_array[k] tile. lda >= m.
///
/// @param[out] dAT_array
///     Array in GPU memory of dimension batch_count, containing pointers to
///     matrices, where each dAT_array[k] is a rectangular m-by-n matrix stored in an
///     ldat-by-m array in GPU memory.
///     On output, each dAT_array[k] is the transpose of dA_array[k].
///
/// @param[in] lda
///     Leading dimension of each dAT_array[k] tile. ldat >= n.
///
/// @param[in] batch_count
///     Size of Aarray. batch_count >= 0.
///
/// @param[in] stream
///     CUDA stream to execute in.
///
template <typename scalar_t, int NX>
void transpose_batch(
    int64_t m, int64_t n,
    scalar_t **dA_array,  int64_t lda,
    scalar_t **dAT_array, int64_t ldat,
    int64_t batch_count,
    hipStream_t stream)
{
    if ((m <= 0) || (n <= 0))
        return;
    assert(lda >= m);
    assert(ldat >= n);

    int mt = ceildiv( m, int64_t(NB) );
    assert(mt <= 65535);                // CUDA limitation
    int nt = ceildiv( n, int64_t(NB) );
    assert(nt <= 65535);                // CUDA limitation
    assert(batch_count <= 2147483647);  // CUDA limitation, 2^31 - 1

    dim3 grid( uint(batch_count), mt, nt );
    dim3 threads( NX, NY, 1 );
    transpose_batch_kernel<scalar_t, NX><<< grid, threads, 0, stream >>>
        ( m, n, dA_array, lda, dAT_array, ldat );

    // check that launch succeeded (could still have async errors)
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::exception();
    }
}

//------------------------------------------------------------------------------
// Explicit instantiations.
template
void transpose(
    int64_t n,
    float* A, int64_t lda,
    hipStream_t stream);

template
void transpose(
    int64_t n,
    double* A, int64_t lda,
    hipStream_t stream);

template
void transpose(
    int64_t n,
    hipFloatComplex* A, int64_t lda,
    hipStream_t stream);

template
void transpose(
    int64_t n,
    hipDoubleComplex* A, int64_t lda,
    hipStream_t stream);

// ----------------------------------------
template
void transpose_batch(
    int64_t n,
    float** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

template
void transpose_batch(
    int64_t n,
    double** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

template
void transpose_batch(
    int64_t n,
    hipFloatComplex** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);

template
void transpose_batch(
    int64_t n,
    hipDoubleComplex** Aarray, int64_t lda,
    int64_t batch_count,
    hipStream_t stream);


// ----------------------------------------
template<>
void transpose(
    int64_t m, int64_t n,
    float* dA,  int64_t lda,
    float* dAT, int64_t ldat,
    hipStream_t stream)
{
    transpose<float,32>(
        m, n,
        dA,  lda,
        dAT, ldat,
        stream);
}

template<>
void transpose(
    int64_t m, int64_t n,
    double* dA,  int64_t lda,
    double* dAT, int64_t ldat,
    hipStream_t stream)
{
    transpose<double,32>(
        m, n,
        dA,  lda,
        dAT, ldat,
        stream);
}

template<>
void transpose(
    int64_t m, int64_t n,
    hipFloatComplex* dA,  int64_t lda,
    hipFloatComplex* dAT, int64_t ldat,
    hipStream_t stream)
{
    transpose<hipFloatComplex,32>(
        m, n,
        dA,  lda,
        dAT, ldat,
        stream);
}

template<>
void transpose(
    int64_t m, int64_t n,
    hipDoubleComplex* dA,  int64_t lda,
    hipDoubleComplex* dAT, int64_t ldat,
    hipStream_t stream)
{
    transpose<hipDoubleComplex,16>(
        m, n,
        dA,  lda,
        dAT, ldat,
        stream);
}

// ----------------------------------------
template<>
void transpose_batch(
    int64_t m, int64_t n,
    float **dA_array,  int64_t lda,
    float **dAT_array, int64_t ldat,
    int64_t batch_count,
    hipStream_t stream)
{
    transpose_batch<float,32>(
        m, n,
        dA_array,  lda,
        dAT_array, ldat,
        batch_count,
        stream);
}

template<>
void transpose_batch(
    int64_t m, int64_t n,
    double **dA_array,  int64_t lda,
    double **dAT_array, int64_t ldat,
    int64_t batch_count,
    hipStream_t stream)
{
    transpose_batch<double,32>(
        m, n,
        dA_array,  lda,
        dAT_array, ldat,
        batch_count,
        stream);
}

template<>
void transpose_batch(
    int64_t m, int64_t n,
    hipFloatComplex **dA_array,  int64_t lda,
    hipFloatComplex **dAT_array, int64_t ldat,
    int64_t batch_count,
    hipStream_t stream)
{
    transpose_batch<hipFloatComplex,32>(
        m, n,
        dA_array,  lda,
        dAT_array, ldat,
        batch_count,
        stream);
}

template<>
void transpose_batch(
    int64_t m, int64_t n,
    hipDoubleComplex **dA_array,  int64_t lda,
    hipDoubleComplex **dAT_array, int64_t ldat,
    int64_t batch_count,
    hipStream_t stream)
{
    transpose_batch<hipDoubleComplex,16>(
        m, n,
        dA_array,  lda,
        dAT_array, ldat,
        batch_count,
        stream);
}

} // namespace device
} // namespace slate
